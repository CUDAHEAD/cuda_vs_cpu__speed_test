
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>



//#define T 64//1024
//#define N T*10//4096*T
#define NN 200

/*// Kernel function to add the elements of two arrays
__global__ void add_cuda(int *x, int *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < N; i += stride)
    y[i] = x[i] + y[i];
}*/


__global__ void add_cuda_good(int *x,int *y)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  for (int i = 0; i< NN ; i++) {
  y[bid*blockDim.x + tid ] += x[bid*blockDim.x + tid]; 
  y[bid*blockDim.x + tid ] *= 2;
  y[bid*blockDim.x + tid ] += bid*blockDim.x + tid;
  y[bid*blockDim.x + tid ] += 3;
  y[bid*blockDim.x + tid ] += x[bid*blockDim.x + tid]; 
  y[bid*blockDim.x + tid ] *= 2;
  y[bid*blockDim.x + tid ] += bid*blockDim.x + tid;
  y[bid*blockDim.x + tid ] += 3;
  y[bid*blockDim.x + tid ] += x[bid*blockDim.x + tid]; 
  y[bid*blockDim.x + tid ] *= 2;
  y[bid*blockDim.x + tid ] += bid*blockDim.x + tid;
  y[bid*blockDim.x + tid ] += 3;
   }
  
}


/*__global__ void add_cuda_bad(int *x,int *y)
{
  __shared__ int s_x[T];
  __shared__ int s_y[T];
  int tid = threadIdx.x;
  int i = blockDim.x * blockIdx.x + threadIdx.x;
//  if (!threadIdx.x) 
  {
    s_x[tid] = x[i];
    s_y[tid] = y[i];
   // memcpy(s_x,x+blockDim.x * blockIdx.x,T * sizeof(int));
   // memcpy(s_y,y+blockDim.x * blockIdx.x,T * sizeof(int));
  }
  __syncthreads();

  s_y[threadIdx.x] += s_x[threadIdx.x];


 // __syncthreads();
 //if (!threadIdx.x)
 {
   y[i] = s_y[tid];
  //  memcpy(y+blockDim.x * blockIdx.x,s_y,T * sizeof(int));
  }
}
*/


void add_cpu_bad(int *x ,int *y, int size)
{
   for (int i=0; i< size; i++){
    for (int i = 0; i< NN; i++) {
	y[i] += x[i];
	y[i] *= 2;
	y[i] += i;
	y[i] += 3;
	y[i] += x[i];
	y[i] *= 2;
	y[i] += i;
	y[i] += 3;
	y[i] += x[i];
	y[i] *= 2;
	y[i] += i;
	y[i] += 3;
     }
   }
}

void print_1D_arr(const char *text,int arr[], int size)
{
   
   if (text == NULL) printf("\n");
   else printf("--%s--\n",text);
   for (int i=0;i<size;i++)
   {
	printf(":%d:",arr[i]);
   }
   printf("\n");
}

int64_t timespecDiff(struct timespec *timeA_p, struct timespec *timeB_p)
{
  return ((timeA_p->tv_sec * 1000000000) + timeA_p->tv_nsec) -
           ((timeB_p->tv_sec * 1000000000) + timeB_p->tv_nsec);
}

int64_t timeDiffSec(struct timespec *timeA_p, struct timespec *timeB_p)
{
  return timeA_p->tv_sec - timeB_p->tv_sec ;
}

void arr_init(int *x,int *y, int N)
{
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 10;
    y[i] = 2;
  }
}


int main(int argc, char** argv)
{

  struct timespec start, end;
  int *x,*y;
  uint64_t timeElapsedGPU;
  uint64_t timeElapsedCPU;
  int N,T;

  sscanf(argv[1] ,"%d", &N);
  sscanf(argv[2], "%d", &T);
  printf("T=%d,N=%d\n",T,N);

clock_gettime(CLOCK_MONOTONIC, &start);
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(int));
  hipMallocManaged(&y, N*sizeof(int));
 
clock_gettime(CLOCK_MONOTONIC, &end);

  timeElapsedGPU = timespecDiff(&end, &start);
  printf("\n\n\n  timeElapsed for init = %d\n",timeElapsedGPU);

  arr_init(x,y,N);
  print_1D_arr("CUDA:Input",x,10);
  //int blockSize = 1;//4;//256;
  int blockSize = T;//4;//256;
  //int numBlocks = 1;//(N + blockSize - 1) / blockSize;
  int numBlocks = N/blockSize;//(N + blockSize - 1) / blockSize;
  printf(" numBlocks=%d, blockSize=%d\n", numBlocks, blockSize);
  //add_cuda<<<numBlocks, blockSize>>>( x, y);
 clock_gettime(CLOCK_MONOTONIC, &start);  
add_cuda_good<<<numBlocks, blockSize>>>( x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
clock_gettime(CLOCK_MONOTONIC, &end);

  timeElapsedGPU = timespecDiff(&end, &start);
  printf("\n\n\n  timeElapsed GPU = %d\n",timeElapsedGPU);
  printf("\n  Time Diff Sec GPU = %d\n",timeDiffSec(&end,&start));
  print_1D_arr("CUDA:Output",y,10);
  printf("\n\n\n----Final check:%d\n", y[N-1]);

  arr_init(x,y,N);
  print_1D_arr("CUDA:Input",x,10);

  clock_gettime(CLOCK_MONOTONIC, &start);

  // Some code I am interested in measuring 
  add_cpu_bad(x,y,N);

  clock_gettime(CLOCK_MONOTONIC, &end);

  timeElapsedCPU = timespecDiff(&end, &start);
  printf("\n\n\n  timeElapsed CPU= %d ratio:%f\n",timeElapsedCPU, (float)timeElapsedCPU/timeElapsedGPU);
  printf("\n  Time Diff Sec CPU = %d\n",timeDiffSec(&end,&start));

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}
