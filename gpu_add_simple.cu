
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>



#define NN 200


__global__ void add_cuda_good(int *x,int *y)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  for (int i = 0; i< NN ; i++) {
  y[bid*blockDim.x + tid ] += x[bid*blockDim.x + tid]; 
  y[bid*blockDim.x + tid ] *= 2;
  y[bid*blockDim.x + tid ] += bid*blockDim.x + tid;
  y[bid*blockDim.x + tid ] += 3;
  y[bid*blockDim.x + tid ] += x[bid*blockDim.x + tid]; 
  y[bid*blockDim.x + tid ] *= 2;
  y[bid*blockDim.x + tid ] += bid*blockDim.x + tid;
  y[bid*blockDim.x + tid ] += 3;
  y[bid*blockDim.x + tid ] += x[bid*blockDim.x + tid]; 
  y[bid*blockDim.x + tid ] *= 2;
  y[bid*blockDim.x + tid ] += bid*blockDim.x + tid;
  y[bid*blockDim.x + tid ] += 3;
   }
  
}


void add_cpu_bad(int *x ,int *y, int size)
{
   for (int i=0; i< size; i++){
    for (int i = 0; i< NN; i++) {
	y[i] += x[i];
	y[i] *= 2;
	y[i] += i;
	y[i] += 3;
	y[i] += x[i];
	y[i] *= 2;
	y[i] += i;
	y[i] += 3;
	y[i] += x[i];
	y[i] *= 2;
	y[i] += i;
	y[i] += 3;
     }
   }
}

void print_1D_arr(const char *text,int arr[], int size)
{
   
   if (text == NULL) printf("\n");
   else printf("--%s--\n",text);
   for (int i=0;i<size;i++)
   {
	printf(":%d:",arr[i]);
   }
   printf("\n");
}

int64_t timespecDiff(struct timespec *timeA_p, struct timespec *timeB_p)
{
  return ((timeA_p->tv_sec * 1000000000) + timeA_p->tv_nsec) -
           ((timeB_p->tv_sec * 1000000000) + timeB_p->tv_nsec);
}

int64_t timeDiffSec(struct timespec *timeA_p, struct timespec *timeB_p)
{
  return timeA_p->tv_sec - timeB_p->tv_sec ;
}

void arr_init(int *x,int *y, int N)
{
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 10;
    y[i] = 2;
  }
}


int main(int argc, char** argv)
{

  struct timespec start, end;
  int *x,*y;
  uint64_t timeElapsedGPU;
  uint64_t timeElapsedCPU;
  int N,T;

  sscanf(argv[1] ,"%d", &N);
  sscanf(argv[2], "%d", &T);

clock_gettime(CLOCK_MONOTONIC, &start);
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(int));
  hipMallocManaged(&y, N*sizeof(int));
 
clock_gettime(CLOCK_MONOTONIC, &end);

  timeElapsedGPU = timespecDiff(&end, &start);
  printf("\n\n\n  timeElapsed for init = %d\n",timeElapsedGPU);

  arr_init(x,y,N);
//  print_1D_arr("CUDA:Input",x,10);
  int blockSize = T;
  int numBlocks = N/blockSize;
  printf(" numBlocks=%d, blockSize=%d\n", numBlocks, blockSize);
clock_gettime(CLOCK_MONOTONIC, &start);  
  add_cuda_good<<<numBlocks, blockSize>>>( x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
clock_gettime(CLOCK_MONOTONIC, &end);

  timeElapsedGPU = timespecDiff(&end, &start);
  printf("\n\n\n  timeElapsed GPU = %d\n",timeElapsedGPU);
  printf("\n  Time Diff Sec GPU = %d\n",timeDiffSec(&end,&start));
//  print_1D_arr("CUDA:Output",y,10);
//  printf("\n\n\n----Final check:%d\n", y[N-1]);

  arr_init(x,y,N);
//  print_1D_arr("CUDA:Input",x,10);

clock_gettime(CLOCK_MONOTONIC, &start);

  // Some code I am interested in measuring 
  add_cpu_bad(x,y,N);

clock_gettime(CLOCK_MONOTONIC, &end);

  timeElapsedCPU = timespecDiff(&end, &start);
  printf("\n\n\n  timeElapsed CPU= %d ratio:%f\n",timeElapsedCPU, (float)timeElapsedCPU/timeElapsedGPU);
  printf("\n  Time Diff Sec CPU = %d\n",timeDiffSec(&end,&start));

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}
